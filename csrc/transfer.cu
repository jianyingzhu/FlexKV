#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) <2025> NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "transfer.cuh"

namespace flexkv {

#define FLOAT4_PTR(ptr) reinterpret_cast<float4 *>(ptr)

__global__ void transfer_kv_blocks_kernel(
    int num_blocks, int start_layer_id, int num_layers, int64_t *gpu_block_ids,
    int64_t **k_gpu_layer_ptrs, int64_t **v_gpu_layer_ptrs,
    int64_t gpu_block_stride, int64_t gpu_startoff_inside_chunks,
    int64_t *cpu_block_ids, int64_t *cpu_ptr, int64_t cpu_kv_stride,
    int64_t cpu_layer_stride, int64_t cpu_block_stride,
    int64_t cpu_startoff_inside_chunks, int64_t copy_size, bool is_mla,
    bool is_host_to_device) {
  int kv_dim = is_mla ? 1 : 2;
  int num_chunks = num_layers * kv_dim * num_blocks;
  int64_t copy_size_in_float4 = copy_size * sizeof(int64_t) / sizeof(float4);

  for (int chunk_idx = blockIdx.x; chunk_idx < num_chunks;
       chunk_idx += gridDim.x) {
    int layer_idx = chunk_idx / (num_blocks * kv_dim);
    int kv_idx = (chunk_idx % (num_blocks * kv_dim)) / num_blocks;
    int gpu_block_idx = gpu_block_ids[chunk_idx % num_blocks];
    int cpu_block_idx = cpu_block_ids[chunk_idx % num_blocks];

    int64_t *cpu_chunk_ptr =
        cpu_ptr + (layer_idx + start_layer_id) * cpu_layer_stride +
        kv_idx * cpu_kv_stride + cpu_block_idx * cpu_block_stride +
        cpu_startoff_inside_chunks;
    int64_t **gpu_layer_ptrs =
        (kv_idx == 0) ? k_gpu_layer_ptrs : v_gpu_layer_ptrs;
    int64_t *gpu_chunk_ptr = gpu_layer_ptrs[layer_idx] +
                             gpu_block_idx * gpu_block_stride +
                             gpu_startoff_inside_chunks;

    int64_t *src_chunk_ptr = is_host_to_device ? cpu_chunk_ptr : gpu_chunk_ptr;
    int64_t *dst_chunk_ptr = is_host_to_device ? gpu_chunk_ptr : cpu_chunk_ptr;

    for (int64_t idx = threadIdx.x; idx < copy_size_in_float4;
         idx += blockDim.x) {
      float4 element = __ldg(&FLOAT4_PTR(src_chunk_ptr)[idx]);
      FLOAT4_PTR(dst_chunk_ptr)[idx] = element;
    }
  }
}

void transfer_kv_blocks(
    int num_blocks, int start_layer_id, int num_layers, int64_t *gpu_block_ids,
    void **k_gpu_layer_ptrs, void **v_gpu_layer_ptrs,
    int64_t gpu_block_stride_in_bytes, int64_t gpu_startoff_inside_chunks,
    int64_t *cpu_block_ids, void *cpu_ptr, int64_t cpu_kv_stride_in_bytes,
    int64_t cpu_layer_stride_in_bytes, int64_t cpu_block_stride_in_bytes,
    int64_t cpu_startoff_inside_chunks, int64_t chunk_size_in_bytes,
    hipStream_t stream, int transfer_sms, bool is_host_to_device,
    bool use_ce_transfer, bool is_mla) {
  int block_size = 128;
  static int max_blocks_per_sm = -1;
  if (max_blocks_per_sm == -1) {
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_blocks_per_sm, transfer_kv_blocks_kernel, block_size, 0);
  }

  if (transfer_sms == -1) {
    transfer_sms = 4;
  }

  int block_count = transfer_sms * max_blocks_per_sm;

  int64_t **k_gpu_layer_ptrs_int64 =
      reinterpret_cast<int64_t **>(k_gpu_layer_ptrs);
  int64_t **v_gpu_layer_ptrs_int64 =
      reinterpret_cast<int64_t **>(v_gpu_layer_ptrs);
  int64_t *cpu_ptr_int64 = reinterpret_cast<int64_t *>(cpu_ptr);
  int64_t cpu_kv_stride_int64 = cpu_kv_stride_in_bytes / sizeof(int64_t);
  int64_t gpu_block_stride_int64 = gpu_block_stride_in_bytes / sizeof(int64_t);
  int64_t cpu_block_stride_int64 = cpu_block_stride_in_bytes / sizeof(int64_t);
  int64_t cpu_layer_stride_int64 = cpu_layer_stride_in_bytes / sizeof(int64_t);
  int64_t cpu_startoff_inside_chunks_int64 =
      cpu_startoff_inside_chunks / sizeof(int64_t);
  int64_t gpu_startoff_inside_chunks_int64 =
      gpu_startoff_inside_chunks / sizeof(int64_t);
  int64_t chunk_size_in_int64 = chunk_size_in_bytes / sizeof(int64_t);

  dim3 blockDim(block_size);
  dim3 gridDim(block_count);
  if (use_ce_transfer) {
    for (int i = 0; i < num_layers; i++) {
      int kv_dim = is_mla ? 1 : 2;
      for (int j = 0; j < kv_dim; j++) {
        for (int k = 0; k < num_blocks; k++) {
          int64_t gpu_block_idx = gpu_block_ids[k];
          int64_t cpu_block_idx = cpu_block_ids[k];
          int64_t *cpu_chunk_ptr =
              cpu_ptr_int64 + (i + start_layer_id) * cpu_layer_stride_int64 +
              j * cpu_kv_stride_int64 + cpu_block_idx * cpu_block_stride_int64 +
              cpu_startoff_inside_chunks_int64;
          int64_t **gpu_layer_ptrs_int64 =
              (j == 0) ? k_gpu_layer_ptrs_int64 : v_gpu_layer_ptrs_int64;
          int64_t *gpu_chunk_ptr = gpu_layer_ptrs_int64[i] +
                                   gpu_block_idx * gpu_block_stride_int64 +
                                   gpu_startoff_inside_chunks_int64;

          if (is_host_to_device) {
            hipMemcpyAsync(gpu_chunk_ptr, cpu_chunk_ptr, chunk_size_in_bytes,
                            hipMemcpyHostToDevice, stream);
          } else {
            hipMemcpyAsync(cpu_chunk_ptr, gpu_chunk_ptr, chunk_size_in_bytes,
                            hipMemcpyDeviceToHost, stream);
          }
        }
      }
    }
  } else {
    transfer_kv_blocks_kernel<<<gridDim, blockDim, 0, stream>>>(
        num_blocks, start_layer_id, num_layers, gpu_block_ids,
        k_gpu_layer_ptrs_int64, v_gpu_layer_ptrs_int64, gpu_block_stride_int64,
        gpu_startoff_inside_chunks_int64, cpu_block_ids, cpu_ptr_int64,
        cpu_kv_stride_int64, cpu_layer_stride_int64, cpu_block_stride_int64,
        cpu_startoff_inside_chunks_int64, chunk_size_in_int64, is_mla,
        is_host_to_device);
  }
  hipStreamSynchronize(stream);
}

} // namespace flexkv
